#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: DeepLearningNetwork.cu
//
// GPU Coder version                    : 1.0
// CUDA/C/C++ source code generated on  : 07-Jan-2018 02:03:56
//

// Include Files
#include "rt_nonfinite.h"
#include "detect_lane.h"
#include "DeepLearningNetwork.h"

// Type Definitions
#include "cnn_api.hpp"

// Function Declarations
static __global__ void c_DeepLearningNetwork_predict_k(const real32_T *inputdata,
  real32_T *inputT);
static __global__ void d_DeepLearningNetwork_predict_k(real32_T *out, real32_T
  *outT);

// Function Definitions

//
// Arguments    : uint3 blockArg
//                uint3 gridArg
//                const real32_T *inputdata
//                real32_T *inputT
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void c_DeepLearningNetwork_predict_k
  (const real32_T *inputdata, real32_T *inputT)
{
  uint32_T threadId;
  int32_T i1;
  int32_T i2;
  int32_T p;
  ;
  ;
  threadId = ((((gridDim.x * gridDim.y * blockIdx.z + gridDim.x * blockIdx.y) +
                blockIdx.x) * (blockDim.x * blockDim.y * blockDim.z) +
               threadIdx.z * blockDim.x * blockDim.y) + threadIdx.y * blockDim.x)
    + threadIdx.x;
  p = (int32_T)(threadId / 51529U);
  i2 = (int32_T)((threadId - (uint32_T)p * 51529U) / 227U);
  i1 = (int32_T)((threadId - (uint32_T)i2 * 227U) - (uint32_T)p * 51529U);
  if (((int32_T)((!(int32_T)(i1 >= 227)) && (!(int32_T)(i2 >= 227)))) &&
      (!(int32_T)(p >= 3))) {
    inputT[(i1 + 227 * i2) + 51529 * p] = inputdata[(i2 + 227 * i1) + 51529 * p];
  }
}

//
// Arguments    : MWCNNLayer *this
// Return Type  : void
//

//
// Arguments    : MWConvLayer *this
//                MWCNNLayer *previousLayer
//                int32_T FilterSizeH
//                int32_T FilterSizeW
//                int32_T NumChannels
//                int32_T NumFilters
//                int32_T StrideH
//                int32_T StrideW
//                int32_T PaddingH
//                int32_T PaddingW
//                int32_T NumGroups
// Return Type  : void
//

//
// Arguments    : MWFCLayer *this
//                MWCNNLayer *previousLayer
//                int32_T InputSize
//                int32_T OutputSize
// Return Type  : void
//

//
// Arguments    : MWInputLayer *this
//                int32_T n
//                int32_T h
//                int32_T w
//                int32_T c
//                boolean_T withAvg
// Return Type  : void
//

//
// Arguments    : MWMaxPoolingLayer *this
//                MWCNNLayer *previousLayer
//                int32_T PoolSizeH
//                int32_T PoolSizeW
//                int32_T StrideH
//                int32_T StrideW
//                int32_T PaddingH
//                int32_T PaddingW
// Return Type  : void
//

//
// Arguments    : MWNormLayer *this
//                MWCNNLayer *previousLayer
//                int32_T WindowSize
//                real_T Alpha
//                real_T Beta
//                real_T K
// Return Type  : void
//

//
// Arguments    : MWOutputLayer *this
//                MWCNNLayer *previousLayer
// Return Type  : void
//

//
// Arguments    : MWPassthroughLayer *this
//                MWCNNLayer *previousLayer
// Return Type  : void
//

//
// Arguments    : MWReLULayer *this
//                MWCNNLayer *previousLayer
// Return Type  : void
//

//
// Arguments    : MWCNNLayer *this
//                real32_T **handle
// Return Type  : void
//

//
// Arguments    : uint3 blockArg
//                uint3 gridArg
//                real32_T *out
//                real32_T *outT
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void d_DeepLearningNetwork_predict_k
  (real32_T *out, real32_T *outT)
{
  int32_T i1;
  ;
  ;
  i1 = (int32_T)(((((gridDim.x * gridDim.y * blockIdx.z + gridDim.x * blockIdx.y)
                    + blockIdx.x) * (blockDim.x * blockDim.y * blockDim.z) +
                   threadIdx.z * blockDim.x * blockDim.y) + threadIdx.y *
                  blockDim.x) + threadIdx.x);
  if (!(int32_T)(i1 >= 6)) {
    outT[i1] = out[i1];
  }
}

//
// Arguments    : MWCNNLayer *this
// Return Type  : void
//

//
// Arguments    : MWInputLayer *this
//                const char * fileName
// Return Type  : void
//

//
// Arguments    : MWConvLayer *this
//                const char * fileName
// Return Type  : void
//

//
// Arguments    : MWFCLayer *this
//                const char * fileName
// Return Type  : void
//

//
// Arguments    : MWFCLayer *this
//                const char * fileName
// Return Type  : void
//

//
// Arguments    : MWConvLayer *this
//                const char * fileName
// Return Type  : void
//

//
// Arguments    : MWCNNLayer *this
// Return Type  : void
//

//
// Arguments    : MWCNNLayer *this
//                hipblasHandle_t *handle
// Return Type  : void
//

//
// Arguments    : MWCNNLayer *this
//                hipdnnHandle_t *handle
// Return Type  : void
//

//
// Arguments    : MWCNNLayer *this
//                real32_T *handle
// Return Type  : void
//

//
// Arguments    : c_lanenet *obj
//                const real32_T inputdata[154587]
//                real32_T outT[6]
// Return Type  : void
//
void DeepLearningNetwork_predict(c_lanenet *obj, const real32_T inputdata[154587],
  real32_T outT[6])
{
  real32_T *gpu_inputT;
  real32_T *gpu_out;
  real32_T *gpu_inputdata;
  real32_T *gpu_outT;
  hipMalloc(&gpu_outT, 24ULL);
  hipMalloc(&gpu_out, 24ULL);
  hipMalloc(&gpu_inputT, 618348ULL);
  hipMalloc(&gpu_inputdata, 618348ULL);
  hipMemcpy((void *)gpu_inputdata, (void *)&inputdata[0], 618348ULL,
             hipMemcpyHostToDevice);
  c_DeepLearningNetwork_predict_k<<<dim3(302U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (gpu_inputdata, gpu_inputT);
  hipMemcpy(obj->inputData, gpu_inputT, 154587ULL * sizeof(real32_T),
             hipMemcpyDeviceToDevice);
  obj->predict();
  hipMemcpy(gpu_out, obj->outputData, 6ULL * sizeof(real32_T),
             hipMemcpyDeviceToDevice);
  d_DeepLearningNetwork_predict_k<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (gpu_out, gpu_outT);
  hipMemcpy((void *)&outT[0], (void *)gpu_outT, 24ULL, hipMemcpyDeviceToHost);
  hipFree(gpu_inputdata);
  hipFree(gpu_inputT);
  hipFree(gpu_out);
  hipFree(gpu_outT);
}

//
// Arguments    : c_lanenet *obj
// Return Type  : void
//
void DeepLearningNetwork_setup(c_lanenet *obj)
{
  obj->setup();
  obj->batchSize = 1;
}

//
// Arguments    : c_lanenet *this
// Return Type  : void
//
void c_lanenet::cleanup()
{
  int32_T idx;
  for (idx = 0; idx < 23; idx++) {
    this->layers[idx]->cleanup();
  }

  if (this->workSpace) {
    hipFree(this->workSpace);
  }

  if (this->cublasHandle) {
    hipblasDestroy(*this->cublasHandle);
  }

  if (this->cudnnHandle) {
    hipdnnDestroy(*this->cudnnHandle);
  }
}

//
// Arguments    : c_lanenet *this
// Return Type  : void
//
c_lanenet::c_lanenet()
{
  this->numLayers = 23;
  this->cublasHandle = 0;
  this->cudnnHandle = 0;
  this->workSpace = 0;
  this->layers[0] = new MWInputLayer;
  this->layers[1] = new MWConvLayer;
  this->layers[2] = new MWReLULayer;
  this->layers[3] = new MWNormLayer;
  this->layers[4] = new MWMaxPoolingLayer;
  this->layers[5] = new MWConvLayer;
  this->layers[6] = new MWReLULayer;
  this->layers[7] = new MWNormLayer;
  this->layers[8] = new MWMaxPoolingLayer;
  this->layers[9] = new MWConvLayer;
  this->layers[10] = new MWReLULayer;
  this->layers[11] = new MWConvLayer;
  this->layers[12] = new MWReLULayer;
  this->layers[13] = new MWConvLayer;
  this->layers[14] = new MWReLULayer;
  this->layers[15] = new MWMaxPoolingLayer;
  this->layers[16] = new MWFCLayer;
  this->layers[17] = new MWReLULayer;
  this->layers[18] = new MWPassthroughLayer;
  this->layers[19] = new MWFCLayer;
  this->layers[20] = new MWReLULayer;
  this->layers[21] = new MWFCLayer;
  this->layers[22] = new MWOutputLayer;
}

//
// Arguments    : c_lanenet *this
// Return Type  : void
//
c_lanenet::~c_lanenet()
{
  int32_T idx;
  this->cleanup();
  for (idx = 0; idx < 23; idx++) {
    delete this->layers[idx];
  }
}

//
// Arguments    : c_lanenet *this
// Return Type  : void
//
void c_lanenet::predict()
{
  int32_T idx;
  for (idx = 0; idx < 23; idx++) {
    this->layers[idx]->predict();
  }
}

//
// Arguments    : c_lanenet *this
// Return Type  : void
//
void c_lanenet::setup()
{
  int32_T idx_handles;
  int32_T idx_ws;
  this->cublasHandle = new hipblasHandle_t;
  hipblasCreate(this->cublasHandle);
  this->cudnnHandle = new hipdnnHandle_t;
  hipdnnCreate(this->cudnnHandle);
  for (idx_handles = 0; idx_handles < 23; idx_handles++) {
    this->layers[idx_handles]->setCublasHandle(this->cublasHandle);
    this->layers[idx_handles]->setCudnnHandle(this->cudnnHandle);
  }

  this->layers[0]->createInputLayer(1, 227, 227, 3, 1);
  this->layers[0]->loadAvg("./codegen/lib/detect_lane/cnn_lanenet_avg");
  this->layers[1]->createConvLayer(this->layers[0], 11, 11, 3, 96, 4, 4, 0, 0, 1);
  this->layers[1]->loadWeights(
    "./codegen/lib/detect_lane/cnn_lanenet_conv1_w");
  this->layers[1]->loadBias("./codegen/lib/detect_lane/cnn_lanenet_conv1_b");
  this->layers[2]->createReLULayer(this->layers[1]);
  this->layers[3]->createNormLayer(this->layers[2], 5, 0.0001, 0.75, 1.0);
  this->layers[4]->createMaxPoolingLayer(this->layers[3], 3, 3, 2, 2, 0, 0);
  this->layers[5]->createConvLayer(this->layers[4], 5, 5, 48, 128, 1, 1, 2, 2, 2);
  this->layers[5]->loadWeights(
    "./codegen/lib/detect_lane/cnn_lanenet_conv2_w");
  this->layers[5]->loadBias("./codegen/lib/detect_lane/cnn_lanenet_conv2_b");
  this->layers[6]->createReLULayer(this->layers[5]);
  this->layers[7]->createNormLayer(this->layers[6], 5, 0.0001, 0.75, 1.0);
  this->layers[8]->createMaxPoolingLayer(this->layers[7], 3, 3, 2, 2, 0, 0);
  this->layers[9]->createConvLayer(this->layers[8], 3, 3, 256, 384, 1, 1, 1, 1,
    1);
  this->layers[9]->loadWeights(
    "./codegen/lib/detect_lane/cnn_lanenet_conv3_w");
  this->layers[9]->loadBias("./codegen/lib/detect_lane/cnn_lanenet_conv3_b");
  this->layers[10]->createReLULayer(this->layers[9]);
  this->layers[11]->createConvLayer(this->layers[10], 3, 3, 192, 192, 1, 1, 1, 1,
    2);
  this->layers[11]->loadWeights(
    "./codegen/lib/detect_lane/cnn_lanenet_conv4_w");
  this->layers[11]->loadBias("./codegen/lib/detect_lane/cnn_lanenet_conv4_b");
  this->layers[12]->createReLULayer(this->layers[11]);
  this->layers[13]->createConvLayer(this->layers[12], 3, 3, 192, 128, 1, 1, 1, 1,
    2);
  this->layers[13]->loadWeights(
    "./codegen/lib/detect_lane/cnn_lanenet_conv5_w");
  this->layers[13]->loadBias("./codegen/lib/detect_lane/cnn_lanenet_conv5_b");
  this->layers[14]->createReLULayer(this->layers[13]);
  this->layers[15]->createMaxPoolingLayer(this->layers[14], 3, 3, 2, 2, 0, 0);
  this->layers[16]->createFCLayer(this->layers[15], 9216, 4096);
  this->layers[16]->loadWeights(
    "./codegen/lib/detect_lane/cnn_lanenet_fc6_w");
  this->layers[16]->loadBias("./codegen/lib/detect_lane/cnn_lanenet_fc6_b");
  this->layers[17]->createReLULayer(this->layers[16]);
  this->layers[18]->createPassthroughLayer(this->layers[17]);
  this->layers[19]->createFCLayer(this->layers[18], 4096, 16);
  this->layers[19]->loadWeights(
    "./codegen/lib/detect_lane/cnn_lanenet_fcLane1_w");
  this->layers[19]->loadBias(
    "./codegen/lib/detect_lane/cnn_lanenet_fcLane1_b");
  this->layers[20]->createReLULayer(this->layers[19]);
  this->layers[21]->createFCLayer(this->layers[20], 16, 6);
  this->layers[21]->loadWeights(
    "./codegen/lib/detect_lane/cnn_lanenet_fcLane2_w");
  this->layers[21]->loadBias(
    "./codegen/lib/detect_lane/cnn_lanenet_fcLane2_b");
  this->layers[22]->createOutputLayer(this->layers[21]);
  this->layers[22]->createWorkSpace((&this->workSpace));
  for (idx_ws = 0; idx_ws < 23; idx_ws++) {
    this->layers[idx_ws]->setWorkSpace(this->workSpace);
  }

  this->inputData = this->layers[0]->getData();
  this->outputData = this->layers[22]->getData();
}

//
// File trailer for DeepLearningNetwork.cu
//
// [EOF]
//
