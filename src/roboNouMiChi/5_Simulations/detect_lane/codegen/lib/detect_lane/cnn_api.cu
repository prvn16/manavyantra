#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <stdexcept>
#include <string>
#include <cassert>
#include "cnn_api.hpp"
#ifdef RANDOM
#include <hiprand.h>
 hiprandGenerator_t VCbcPxtPsBLTrHYdEvqn; void 
curand_call_line_file(hiprandStatus_t lteHjcLsItGbVPMQtGDB, const int 
eFaDPmxDdzHlRYSAoMmX, const char *RqCYCrGsNvzKYrRMXbsI) { if (lteHjcLsItGbVPMQtGDB != 
HIPRAND_STATUS_SUCCESS) { printf("%d, line: %d, file: %s\n", lteHjcLsItGbVPMQtGDB, 
eFaDPmxDdzHlRYSAoMmX, RqCYCrGsNvzKYrRMXbsI); exit(EXIT_FAILURE); } }
#endif
 void call_cuda_free(float* mem) { hipError_t lteHjcLsItGbVPMQtGDB = 
hipFree(mem); if (lteHjcLsItGbVPMQtGDB != hipErrorDeinitialized) { 
CUDA_CALL(lteHjcLsItGbVPMQtGDB); } } void cuda_call_line_file(hipError_t 
lteHjcLsItGbVPMQtGDB, const int eFaDPmxDdzHlRYSAoMmX, const char *RqCYCrGsNvzKYrRMXbsI) { if 
(lteHjcLsItGbVPMQtGDB != hipSuccess) { printf("%s, line: %d, file: %s\n", 
hipGetErrorString(lteHjcLsItGbVPMQtGDB), eFaDPmxDdzHlRYSAoMmX, RqCYCrGsNvzKYrRMXbsI); 
exit(EXIT_FAILURE); } } void cudnn_call_line_file(hipdnnStatus_t 
lteHjcLsItGbVPMQtGDB, const int eFaDPmxDdzHlRYSAoMmX, const char *RqCYCrGsNvzKYrRMXbsI) { if 
(lteHjcLsItGbVPMQtGDB != HIPDNN_STATUS_SUCCESS) { 
printf("%s, line: %d, file: %s\n", hipdnnGetErrorString(lteHjcLsItGbVPMQtGDB), 
eFaDPmxDdzHlRYSAoMmX, RqCYCrGsNvzKYrRMXbsI); exit(EXIT_FAILURE); } } const char* 
cublasGetErrorString(hipblasStatus_t lteHjcLsItGbVPMQtGDB) { 
switch(lteHjcLsItGbVPMQtGDB) { case HIPBLAS_STATUS_SUCCESS: return 
"HIPBLAS_STATUS_SUCCESS"; case HIPBLAS_STATUS_NOT_INITIALIZED: return 
"HIPBLAS_STATUS_NOT_INITIALIZED"; case HIPBLAS_STATUS_ALLOC_FAILED: return 
"HIPBLAS_STATUS_ALLOC_FAILED"; case HIPBLAS_STATUS_INVALID_VALUE: return 
"HIPBLAS_STATUS_INVALID_VALUE";  case HIPBLAS_STATUS_ARCH_MISMATCH: return 
"HIPBLAS_STATUS_ARCH_MISMATCH";  case HIPBLAS_STATUS_MAPPING_ERROR: return 
"HIPBLAS_STATUS_MAPPING_ERROR"; case HIPBLAS_STATUS_EXECUTION_FAILED: return 
"HIPBLAS_STATUS_EXECUTION_FAILED";  case HIPBLAS_STATUS_INTERNAL_ERROR: return 
"HIPBLAS_STATUS_INTERNAL_ERROR";  case HIPBLAS_STATUS_NOT_SUPPORTED: return 
"HIPBLAS_STATUS_NOT_SUPPORTED";  case HIPBLAS_STATUS_UNKNOWN: return 
"HIPBLAS_STATUS_UNKNOWN";  } return "unknown error"; } void 
cublas_call_line_file(hipblasStatus_t lteHjcLsItGbVPMQtGDB, const int 
eFaDPmxDdzHlRYSAoMmX, const char *RqCYCrGsNvzKYrRMXbsI) { if (lteHjcLsItGbVPMQtGDB != 
HIPBLAS_STATUS_SUCCESS) { printf("%s, line: %d, file: %s\n", 
cublasGetErrorString(lteHjcLsItGbVPMQtGDB), eFaDPmxDdzHlRYSAoMmX, RqCYCrGsNvzKYrRMXbsI); 
exit(EXIT_FAILURE); } } MWCNNLayer::MWCNNLayer() : sRECVoNNtDdcBOWgDyar(NULL), 
AIXLuRgdeiqpaCehGSYD(1), BRSPqxNffoBYKqpSVHne(0), IbSWJNMuIiKbocfQKqXb(0), 
DqxLTLaJwwgQqmrtCDuu(0), ECTnqgWHyHCHCLBZlffd(0), 
xHViLEwTujGGrPZZgmbF(NULL), vxtNGOWYjhKeBBSzuIMB(NULL), xkUNToJIgvoLoUQuzKRF(NULL), 
OumvfgWXDdmsQaciHMHx(NULL), OwenhowBxTAXHXmJpIKd(NULL), 
QVgVGfoCXYiYXzPhvVPX(0.0), QMgBqCuvjnbWHWiVPEwn(1.0), PtkeOkuClHzhOfpmBevf(-1.0), 
bMAyVFGSPDjmUbziYLAy(0)  { 
CUDNN_CALL(hipdnnCreateTensorDescriptor(getGroupDescriptor())); } 
MWCNNLayer::~MWCNNLayer() { 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(*getGroupDescriptor())); } FILE* 
MWCNNLayer::openBinaryFile(const char* fileName) { FILE* fp = fopen(fileName, 
"rb"); if (!fp) {
#if defined(_WIN32) || defined(_WIN64)
 char delim[] = "\\";
#else
 char delim[] = "/";
#endif
 std::string fileS(fileName); size_t pos = 0; while((pos = fileS.find(delim)) 
!= std::string::npos) { if (pos == (fileS.size() - 1)) { fileS = ""; break; } 
fileS = fileS.substr(pos+1); } if (!fileS.empty()) { fp = fopen(fileS.c_str(), 
"rb");  } if (!fp) { printf("Error! Unable to open file %s\n", fileS.c_str()); 
throw std::runtime_error("Error opening file!!\n"); } } return fp; } 
std::runtime_error MWCNNLayer::getFileOpenError(const char* filename) { const 
std::string message = std::string("Error! Unable to open file ") + 
std::string(filename); return std::runtime_error(message); } void 
MWCNNLayer::createInputLayer(int, int, int, int, bool) { return; } void 
MWCNNLayer::createReLULayer(MWCNNLayer*) { return; } void 
MWCNNLayer::createLeakyReLULayer(MWCNNLayer*, double) { return; } void 
MWCNNLayer::createBatchNormalizationLayer(MWCNNLayer*, double) { return; } void 
MWCNNLayer::createSoftmaxLayer(MWCNNLayer*) { return; } void 
MWCNNLayer::createOutputLayer(MWCNNLayer*) { return; } void 
MWCNNLayer::createFCLayer(MWCNNLayer*, int, int) { return; } void 
MWCNNLayer::createNormLayer(MWCNNLayer*, unsigned, double, double, double) { 
return; } void MWCNNLayer::createAvgPoolingLayer(MWCNNLayer*, int, int, int, 
int, int, int) { return; } void MWCNNLayer::createMaxPoolingLayer(MWCNNLayer*, 
int, int, int, int, int, int) { return; } void 
MWCNNLayer::createConvLayer(MWCNNLayer*, int, int, int, int, int, int, int, 
int, int) { return; } void MWCNNLayer::loadAvg(const char*) { return; } void 
MWCNNLayer::createPassthroughLayer(MWCNNLayer*) { return; } void 
MWCNNLayer::loadWeights(const char*) { return; } void 
MWCNNLayer::loadBias(const char*) { return; } void MWCNNLayer::setName(const 
char* n) { fOpFYwKNwIfWjnPzNuob = n; return; } void 
MWCNNLayer::createWorkSpace(float** wtNPjzxHKNoJIigzXrEl) { 
CUDA_CALL(hipMalloc((void**)wtNPjzxHKNoJIigzXrEl, 
juRPduBvIGpwaZiftkzr)); return; } void MWCNNLayer::setBatchSize(int bs) 
{ AIXLuRgdeiqpaCehGSYD = bs; return; } int MWCNNLayer::getBatchSize() { return 
AIXLuRgdeiqpaCehGSYD; } void MWCNNLayer::setHeight(int ht) { BRSPqxNffoBYKqpSVHne 
= ht; return; } int MWCNNLayer::getHeight() { return BRSPqxNffoBYKqpSVHne; } void 
MWCNNLayer::setWidth(int wd) { IbSWJNMuIiKbocfQKqXb = wd; return; } int 
MWCNNLayer::getWidth() { return IbSWJNMuIiKbocfQKqXb; } void 
MWCNNLayer::setNumInputFeatures(int nif) { DqxLTLaJwwgQqmrtCDuu = nif; 
return; } int MWCNNLayer::getNumInputFeatures() { return 
DqxLTLaJwwgQqmrtCDuu; } void MWCNNLayer::setNumOutputFeatures(int nof) { 
ECTnqgWHyHCHCLBZlffd = nof; return; } int 
MWCNNLayer::getNumOutputFeatures() { return ECTnqgWHyHCHCLBZlffd; } 
void MWCNNLayer::setData(float* out) { xHViLEwTujGGrPZZgmbF = out; return; } float* 
MWCNNLayer::getData() { return xHViLEwTujGGrPZZgmbF; } float** MWCNNLayer::getDataPtr() 
{ return &xHViLEwTujGGrPZZgmbF; } void 
MWCNNLayer::setOutputDescriptor(hipdnnTensorDescriptor_t od) { 
iPqBiFnIJMxelVhQBZex = od; return; } hipdnnTensorDescriptor_t* 
MWCNNLayer::getOutputDescriptor() { return &iPqBiFnIJMxelVhQBZex; } void 
MWCNNLayer::setWorkSpaceSize(size_t wss) { juRPduBvIGpwaZiftkzr = wss; 
return; } size_t* MWCNNLayer::getWorkSpaceSize() { return 
&juRPduBvIGpwaZiftkzr; } void MWCNNLayer::setWorkSpace(float* ws) { 
vxtNGOWYjhKeBBSzuIMB = ws; return; } float* MWCNNLayer::getWorkSpace() { return 
vxtNGOWYjhKeBBSzuIMB; } void MWCNNLayer::setOutput2(float* out2) { 
xkUNToJIgvoLoUQuzKRF = out2; return; } float* MWCNNLayer::getOutput2() { return 
xkUNToJIgvoLoUQuzKRF; } void MWCNNLayer::setGroupDescriptor(hipdnnTensorDescriptor_t 
gd) { XCLDbxHBtWRStETWIkId = gd; return; } hipdnnTensorDescriptor_t* 
MWCNNLayer::getGroupDescriptor() { return &XCLDbxHBtWRStETWIkId; } void 
MWCNNLayer::setCublasHandle(hipblasHandle_t* cub) { OumvfgWXDdmsQaciHMHx = 
cub; return; } hipblasHandle_t* MWCNNLayer::getCublasHandle() { return 
OumvfgWXDdmsQaciHMHx; } void MWCNNLayer::setCudnnHandle(hipdnnHandle_t* 
cud) { OwenhowBxTAXHXmJpIKd = cud; return; } hipdnnHandle_t* 
MWCNNLayer::getCudnnHandle() { return OwenhowBxTAXHXmJpIKd; } float* 
MWCNNLayer::getZeroPtr() { return &QVgVGfoCXYiYXzPhvVPX; } float* 
MWCNNLayer::getOnePtr() { return &QMgBqCuvjnbWHWiVPEwn; } float* 
MWCNNLayer::getNegOnePtr() { return &PtkeOkuClHzhOfpmBevf; } float 
MWCNNLayer::getIsGrouped() { return bMAyVFGSPDjmUbziYLAy; } void 
MWCNNLayer::setIsGrouped(float ig) { bMAyVFGSPDjmUbziYLAy = ig; return; } 
MWCNNLayer* MWCNNLayer::getTop() { return sRECVoNNtDdcBOWgDyar; } void 
MWCNNLayer::setTop(MWCNNLayer* top) { assert(top != NULL); 
assert(sRECVoNNtDdcBOWgDyar== NULL);  sRECVoNNtDdcBOWgDyar = top; } 
MWInputLayer::MWInputLayer() : JgLfgHrHMEMmMYTettJF(NULL) { 
CUDNN_CALL(hipdnnCreateTensorDescriptor(getOutputDescriptor())); 
CUDNN_CALL(hipdnnCreateTensorDescriptor(&KCudOrFMfgCzUPMcdePX)); } 
MWInputLayer::~MWInputLayer() { } void MWInputLayer::createInputLayer(int 
eqOmMKQRpqBqRQCnJmxt, int XVcMnvCXvZpKICKIjgZi, int ugnnrhsgTeWucrMPCJUc, int NldNILHvuQqQPSAHXxdT, bool 
veFyKKHbdqBIvQLYBqfF) { setBatchSize(eqOmMKQRpqBqRQCnJmxt); 
setNumOutputFeatures(NldNILHvuQqQPSAHXxdT); setHeight(XVcMnvCXvZpKICKIjgZi); 
setWidth(ugnnrhsgTeWucrMPCJUc); bUVPfnrJhLfHzOLUUrKk = veFyKKHbdqBIvQLYBqfF; 
setWorkSpaceSize(0); 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(*getOutputDescriptor(), 
HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, getBatchSize(), getNumOutputFeatures(), 
getHeight(), getWidth())); CUDA_CALL(hipMalloc((void**)getDataPtr(), 
sizeof(float)*getBatchSize()*getNumOutputFeatures()*getHeight()*getWidth())); 
if( bUVPfnrJhLfHzOLUUrKk ) { 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(KCudOrFMfgCzUPMcdePX, HIPDNN_TENSOR_NCHW, 
HIPDNN_DATA_FLOAT, 1, getNumOutputFeatures(), getHeight(), getWidth())); 
CUDA_CALL(hipMalloc((void**)&JgLfgHrHMEMmMYTettJF, 
sizeof(float)*getNumOutputFeatures()*getHeight()*getWidth())); }
#ifdef RANDOM
 hiprandGenerateUniform(VCbcPxtPsBLTrHYdEvqn,getData(), getBatchSize()*getNumOutputFeatures()*getHeight()*getWidth());
#endif
 return; } void MWInputLayer::loadAvg(const char* SDWKEQTZaTFZByPlzUDR) { FILE* 
UKtMXCCqdjeyaVHabkxg = openBinaryFile(SDWKEQTZaTFZByPlzUDR); assert(UKtMXCCqdjeyaVHabkxg); int 
fhikqqlnUKCjleVKDqiG = getHeight()*getWidth()*getNumOutputFeatures();  float* 
NDjzAZSYJuWymuKDNZYB = (float*)malloc(sizeof(float)*fhikqqlnUKCjleVKDqiG); 
fread(NDjzAZSYJuWymuKDNZYB, sizeof(float), fhikqqlnUKCjleVKDqiG, UKtMXCCqdjeyaVHabkxg); 
CUDA_CALL(hipMemcpy(JgLfgHrHMEMmMYTettJF, NDjzAZSYJuWymuKDNZYB, 
sizeof(float)*fhikqqlnUKCjleVKDqiG, hipMemcpyHostToDevice)); 
free(NDjzAZSYJuWymuKDNZYB); fclose(UKtMXCCqdjeyaVHabkxg); return; } void 
MWInputLayer::predict() { if ( bUVPfnrJhLfHzOLUUrKk ) 
CUDNN_CALL(hipdnnAddTensor(*getCudnnHandle(), getNegOnePtr(), 
KCudOrFMfgCzUPMcdePX, JgLfgHrHMEMmMYTettJF, getOnePtr(), *getOutputDescriptor(), 
getData())); return; } void MWInputLayer::cleanup() { 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(*getOutputDescriptor())); if 
(getData()) { call_cuda_free(getData()); } if ( bUVPfnrJhLfHzOLUUrKk ) { 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(KCudOrFMfgCzUPMcdePX)); if (JgLfgHrHMEMmMYTettJF) 
{ call_cuda_free(JgLfgHrHMEMmMYTettJF); } } return; } MWConvLayer::MWConvLayer() : 
ugnnrhsgTeWucrMPCJUc(NULL) , KHClOltUSuqFVVErSxVb(NULL) , vFNECEAeLZsYsUxvlgqL(NULL)  { 
CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&OKaRVOctKLlnIyGmjRNW)); 
CUDNN_CALL(hipdnnCreateFilterDescriptor(&UEESbUvbMihFnquvuFij)); 
CUDNN_CALL(hipdnnCreateTensorDescriptor(&MCrRCXUsCsGPMgQbvMOt)); 
CUDNN_CALL(hipdnnCreateTensorDescriptor(getOutputDescriptor())); } 
MWConvLayer::~MWConvLayer() { } void MWConvLayer::createConvLayer(MWCNNLayer* 
jaqKGCwoANNDMHgAsehk, int jhFUWlztBndwjbXwYNaJ, int kNsviQGMPdXzNMRixGWR, int NldNILHvuQqQPSAHXxdT, 
int dJcdBfQQLhIAYHPxwQeg, int HtQBsWTCGEkpylRklilw, int IAlDgIFcchbwRGBSfVfA, int 
GFienSVKLlDQuZeqAdLC, int GeeOVBfQrpMacIFBLKOo, int UdmcwaUkepxfZrpdpcAN) { int 
olKGEIcsxmLSoMhRhEtP, nlIRrOJaFuVaywxOqOyb, npGnQZLrEfVTQnEbwqij, omxlPZbBePZdWaJOBUUG; 
setTop(jaqKGCwoANNDMHgAsehk); setBatchSize(getTop()->getBatchSize()); 
AzTsxYcYjIEJsGQbeYHm = jhFUWlztBndwjbXwYNaJ; BLjrjqvCcCommiXWQLjs = kNsviQGMPdXzNMRixGWR; 
setNumInputFeatures(NldNILHvuQqQPSAHXxdT*UdmcwaUkepxfZrpdpcAN); 
setNumOutputFeatures(dJcdBfQQLhIAYHPxwQeg*UdmcwaUkepxfZrpdpcAN); DSsxcjIrUgZCKZovyNQf = UdmcwaUkepxfZrpdpcAN;
#if (CUDNN_MAJOR <= 5)
 { CUDNN_CALL(hipdnnSetConvolution2dDescriptor(OKaRVOctKLlnIyGmjRNW, 
GFienSVKLlDQuZeqAdLC, GeeOVBfQrpMacIFBLKOo, HtQBsWTCGEkpylRklilw, 
IAlDgIFcchbwRGBSfVfA, 1, 1, HIPDNN_CROSS_CORRELATION));  }
#else
 { CUDNN_CALL(hipdnnSetConvolution2dDescriptor(OKaRVOctKLlnIyGmjRNW, 
GFienSVKLlDQuZeqAdLC, GeeOVBfQrpMacIFBLKOo, HtQBsWTCGEkpylRklilw, 
IAlDgIFcchbwRGBSfVfA, 1, 1, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));  }
#endif
 if( DSsxcjIrUgZCKZovyNQf == 1 ) { 
CUDNN_CALL(hipdnnSetFilter4dDescriptor(UEESbUvbMihFnquvuFij, HIPDNN_DATA_FLOAT, 
HIPDNN_TENSOR_NCHW, getNumOutputFeatures(), getNumInputFeatures(), 
AzTsxYcYjIEJsGQbeYHm, BLjrjqvCcCommiXWQLjs)); 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(MCrRCXUsCsGPMgQbvMOt, HIPDNN_TENSOR_NCHW, 
HIPDNN_DATA_FLOAT, 1, getNumOutputFeatures(), 1, 1)); 
CUDNN_CALL(hipdnnGetConvolution2dForwardOutputDim(OKaRVOctKLlnIyGmjRNW, 
*getTop()->getOutputDescriptor(), UEESbUvbMihFnquvuFij, &olKGEIcsxmLSoMhRhEtP, 
&nlIRrOJaFuVaywxOqOyb, &npGnQZLrEfVTQnEbwqij, &omxlPZbBePZdWaJOBUUG)); 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(*getOutputDescriptor(), 
HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, olKGEIcsxmLSoMhRhEtP, nlIRrOJaFuVaywxOqOyb, 
npGnQZLrEfVTQnEbwqij, omxlPZbBePZdWaJOBUUG)); 
CUDNN_CALL(hipdnnGetConvolutionForwardAlgorithm(*getCudnnHandle(), 
*getTop()->getOutputDescriptor(), UEESbUvbMihFnquvuFij, OKaRVOctKLlnIyGmjRNW, 
*getOutputDescriptor(), HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, 
&NnAKUXChhnRnQmWsknGy)); 
CUDNN_CALL(hipdnnGetConvolutionForwardWorkspaceSize(*getCudnnHandle(), 
*getTop()->getOutputDescriptor(), UEESbUvbMihFnquvuFij, OKaRVOctKLlnIyGmjRNW, 
*getOutputDescriptor(), NnAKUXChhnRnQmWsknGy, getWorkSpaceSize())); } else { if( 
getTop()->getIsGrouped() == 0 ) { 
CUDNN_CALL(hipdnnSetTensor4dDescriptorEx(*getTop()->getGroupDescriptor(), 
HIPDNN_DATA_FLOAT, getTop()->getBatchSize(), 
getTop()->getNumOutputFeatures()/DSsxcjIrUgZCKZovyNQf, getTop()->getHeight(), 
getTop()->getWidth(), 
getTop()->getNumOutputFeatures()*getTop()->getHeight()*getTop()->getWidth(), 
getTop()->getHeight()*getTop()->getWidth(), getTop()->getWidth(), 1)); 
getTop()->setOutput2(getTop()->getData() + 
getTop()->getNumOutputFeatures()/DSsxcjIrUgZCKZovyNQf * getTop()->getHeight() * 
getTop()->getWidth()); getTop()->setIsGrouped(1); } 
CUDNN_CALL(hipdnnSetFilter4dDescriptor(UEESbUvbMihFnquvuFij, HIPDNN_DATA_FLOAT, 
HIPDNN_TENSOR_NCHW, getNumOutputFeatures()/DSsxcjIrUgZCKZovyNQf, 
getNumInputFeatures()/DSsxcjIrUgZCKZovyNQf, AzTsxYcYjIEJsGQbeYHm, 
BLjrjqvCcCommiXWQLjs)); 
CUDNN_CALL(hipdnnGetConvolution2dForwardOutputDim(OKaRVOctKLlnIyGmjRNW, 
*getTop()->getGroupDescriptor(), UEESbUvbMihFnquvuFij, &olKGEIcsxmLSoMhRhEtP, 
&nlIRrOJaFuVaywxOqOyb, &npGnQZLrEfVTQnEbwqij, &omxlPZbBePZdWaJOBUUG)); 
CUDNN_CALL(hipdnnSetTensor4dDescriptorEx(*getGroupDescriptor(), 
HIPDNN_DATA_FLOAT, olKGEIcsxmLSoMhRhEtP, nlIRrOJaFuVaywxOqOyb, npGnQZLrEfVTQnEbwqij, 
omxlPZbBePZdWaJOBUUG, 
nlIRrOJaFuVaywxOqOyb*DSsxcjIrUgZCKZovyNQf*npGnQZLrEfVTQnEbwqij*omxlPZbBePZdWaJOBUUG, 
npGnQZLrEfVTQnEbwqij*omxlPZbBePZdWaJOBUUG, omxlPZbBePZdWaJOBUUG, 1)); 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(*getOutputDescriptor(), 
HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, olKGEIcsxmLSoMhRhEtP, 
nlIRrOJaFuVaywxOqOyb*DSsxcjIrUgZCKZovyNQf, npGnQZLrEfVTQnEbwqij, omxlPZbBePZdWaJOBUUG)); 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(MCrRCXUsCsGPMgQbvMOt, HIPDNN_TENSOR_NCHW, 
HIPDNN_DATA_FLOAT, 1, nlIRrOJaFuVaywxOqOyb*DSsxcjIrUgZCKZovyNQf, 1, 1)); 
CUDNN_CALL(hipdnnGetConvolutionForwardAlgorithm(*getCudnnHandle(), 
*getTop()->getGroupDescriptor(), UEESbUvbMihFnquvuFij, OKaRVOctKLlnIyGmjRNW, 
*getGroupDescriptor(), HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, 
&NnAKUXChhnRnQmWsknGy)); 
CUDNN_CALL(hipdnnGetConvolutionForwardWorkspaceSize(*getCudnnHandle(), 
*getTop()->getGroupDescriptor(), UEESbUvbMihFnquvuFij, OKaRVOctKLlnIyGmjRNW, 
*getGroupDescriptor(), NnAKUXChhnRnQmWsknGy, getWorkSpaceSize())); } 
setHeight(npGnQZLrEfVTQnEbwqij); setWidth(omxlPZbBePZdWaJOBUUG); if( 
*getWorkSpaceSize() < *getTop()->getWorkSpaceSize() ) { 
setWorkSpaceSize(*getTop()->getWorkSpaceSize()); } 
CUDA_CALL(hipMalloc((void**)getDataPtr(), 
sizeof(float)*getBatchSize()*getNumOutputFeatures()*getHeight()*getWidth())); 
CUDA_CALL(hipMalloc((void**)&ugnnrhsgTeWucrMPCJUc, 
sizeof(float)*getNumInputFeatures()/DSsxcjIrUgZCKZovyNQf*getNumOutputFeatures()*AzTsxYcYjIEJsGQbeYHm*BLjrjqvCcCommiXWQLjs)); 
CUDA_CALL(hipMalloc((void**)&KHClOltUSuqFVVErSxVb, sizeof(float)*getNumOutputFeatures()));
#ifdef RANDOM
 hiprandGenerateNormal(VCbcPxtPsBLTrHYdEvqn, ugnnrhsgTeWucrMPCJUc, 
getNumInputFeatures()/DSsxcjIrUgZCKZovyNQf*getNumOutputFeatures()*AzTsxYcYjIEJsGQbeYHm*BLjrjqvCcCommiXWQLjs, 
0, 0.1); hiprandGenerateNormal(VCbcPxtPsBLTrHYdEvqn, KHClOltUSuqFVVErSxVb, 
getNumOutputFeatures(), -0.5, 1);
#endif
 if( DSsxcjIrUgZCKZovyNQf == 2 ) { vFNECEAeLZsYsUxvlgqL = ugnnrhsgTeWucrMPCJUc + 
getNumOutputFeatures()/DSsxcjIrUgZCKZovyNQf * 
getNumInputFeatures()/DSsxcjIrUgZCKZovyNQf * AzTsxYcYjIEJsGQbeYHm * 
BLjrjqvCcCommiXWQLjs; setOutput2(getData() + getNumOutputFeatures() / 2 * 
getHeight() * getWidth()); setIsGrouped(1); } return; } void 
MWConvLayer::predict() { if( DSsxcjIrUgZCKZovyNQf == 1 ) { 
CUDNN_CALL(hipdnnConvolutionForward(*getCudnnHandle(), getOnePtr(), 
*getTop()->getOutputDescriptor(), getTop()->getData(), UEESbUvbMihFnquvuFij, 
ugnnrhsgTeWucrMPCJUc, OKaRVOctKLlnIyGmjRNW, NnAKUXChhnRnQmWsknGy, getWorkSpace(), 
*getWorkSpaceSize(), getZeroPtr(), *getOutputDescriptor(),getData())); 
CUDNN_CALL(hipdnnAddTensor(*getCudnnHandle(), getOnePtr(), 
MCrRCXUsCsGPMgQbvMOt,KHClOltUSuqFVVErSxVb, getOnePtr(), 
*getOutputDescriptor(),getData())); } else { 
CUDNN_CALL(hipdnnConvolutionForward(*getCudnnHandle(), getOnePtr(), 
*(getTop()->getGroupDescriptor()), getTop()->getData(), UEESbUvbMihFnquvuFij, 
ugnnrhsgTeWucrMPCJUc, OKaRVOctKLlnIyGmjRNW, NnAKUXChhnRnQmWsknGy, getWorkSpace(), 
*getWorkSpaceSize(), getZeroPtr(), *getGroupDescriptor(), getData())); 
CUDNN_CALL(hipdnnConvolutionForward(*getCudnnHandle(), getOnePtr(), 
*(getTop()->getGroupDescriptor()), getTop()->getOutput2(), 
UEESbUvbMihFnquvuFij, vFNECEAeLZsYsUxvlgqL, OKaRVOctKLlnIyGmjRNW, NnAKUXChhnRnQmWsknGy, 
getWorkSpace(), *getWorkSpaceSize(), getZeroPtr(), *getGroupDescriptor(), 
getOutput2())); CUDNN_CALL(hipdnnAddTensor(*getCudnnHandle(), getOnePtr(), 
MCrRCXUsCsGPMgQbvMOt,KHClOltUSuqFVVErSxVb, getOnePtr(), 
*getOutputDescriptor(),getData())); } return; } void MWConvLayer::cleanup() { 
CUDNN_CALL(hipdnnDestroyConvolutionDescriptor(OKaRVOctKLlnIyGmjRNW)); 
CUDNN_CALL(hipdnnDestroyFilterDescriptor(UEESbUvbMihFnquvuFij)); if 
(ugnnrhsgTeWucrMPCJUc) { call_cuda_free(ugnnrhsgTeWucrMPCJUc); } 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(MCrRCXUsCsGPMgQbvMOt)); if 
(KHClOltUSuqFVVErSxVb) { call_cuda_free(KHClOltUSuqFVVErSxVb); } 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(*getOutputDescriptor())); if 
(getData()) { call_cuda_free(getData()); } return; } void 
MWConvLayer::loadWeights(const char* SDWKEQTZaTFZByPlzUDR) { FILE* UKtMXCCqdjeyaVHabkxg 
= openBinaryFile(SDWKEQTZaTFZByPlzUDR); assert(UKtMXCCqdjeyaVHabkxg); int 
fhikqqlnUKCjleVKDqiG = 
getNumInputFeatures()/DSsxcjIrUgZCKZovyNQf*getNumOutputFeatures()*AzTsxYcYjIEJsGQbeYHm*BLjrjqvCcCommiXWQLjs; 
 float* NDjzAZSYJuWymuKDNZYB = (float*)malloc(sizeof(float)*fhikqqlnUKCjleVKDqiG); 
fread(NDjzAZSYJuWymuKDNZYB, sizeof(float), fhikqqlnUKCjleVKDqiG, UKtMXCCqdjeyaVHabkxg); if( 
AzTsxYcYjIEJsGQbeYHm != 1 && BLjrjqvCcCommiXWQLjs != 1 ) { float* 
NMMfJylfQjiIUAKhXCJb = 
(float*)malloc(sizeof(float)*AzTsxYcYjIEJsGQbeYHm*BLjrjqvCcCommiXWQLjs); 
for(int k=0; k<fhikqqlnUKCjleVKDqiG/AzTsxYcYjIEJsGQbeYHm/BLjrjqvCcCommiXWQLjs; 
k++) { for(int i=0; i<AzTsxYcYjIEJsGQbeYHm*BLjrjqvCcCommiXWQLjs; i++) 
NMMfJylfQjiIUAKhXCJb[i]=NDjzAZSYJuWymuKDNZYB[k*AzTsxYcYjIEJsGQbeYHm*BLjrjqvCcCommiXWQLjs+i]; 
for(int j=0; j<AzTsxYcYjIEJsGQbeYHm; j++) for(int i=0; 
i<BLjrjqvCcCommiXWQLjs; i++) 
NDjzAZSYJuWymuKDNZYB[k*AzTsxYcYjIEJsGQbeYHm*BLjrjqvCcCommiXWQLjs+j*BLjrjqvCcCommiXWQLjs+i]=NMMfJylfQjiIUAKhXCJb[j+i*AzTsxYcYjIEJsGQbeYHm]; 
} free(NMMfJylfQjiIUAKhXCJb); } CUDA_CALL(hipMemcpy(ugnnrhsgTeWucrMPCJUc, 
NDjzAZSYJuWymuKDNZYB, sizeof(float)*fhikqqlnUKCjleVKDqiG, hipMemcpyHostToDevice));
#if 0
 printf("%s loaded. Size = %d. %f\n", SDWKEQTZaTFZByPlzUDR, fhikqqlnUKCjleVKDqiG, NDjzAZSYJuWymuKDNZYB[0]);
#endif
 free(NDjzAZSYJuWymuKDNZYB); fclose(UKtMXCCqdjeyaVHabkxg); return; } void 
MWConvLayer::loadBias(const char* SDWKEQTZaTFZByPlzUDR) { FILE* UKtMXCCqdjeyaVHabkxg = 
openBinaryFile(SDWKEQTZaTFZByPlzUDR);  assert(UKtMXCCqdjeyaVHabkxg); int fhikqqlnUKCjleVKDqiG 
= getNumOutputFeatures();  float* NDjzAZSYJuWymuKDNZYB = 
(float*)malloc(sizeof(float)*fhikqqlnUKCjleVKDqiG); fread(NDjzAZSYJuWymuKDNZYB, 
sizeof(float), fhikqqlnUKCjleVKDqiG, UKtMXCCqdjeyaVHabkxg); 
CUDA_CALL(hipMemcpy(KHClOltUSuqFVVErSxVb, NDjzAZSYJuWymuKDNZYB, 
sizeof(float)*fhikqqlnUKCjleVKDqiG, hipMemcpyHostToDevice)); 
free(NDjzAZSYJuWymuKDNZYB); fclose(UKtMXCCqdjeyaVHabkxg); return; } 
MWReLULayer::MWReLULayer() { 
CUDNN_CALL(hipdnnCreateActivationDescriptor(&jscBrjkVJyVfMMDjFpgl)); 
CUDNN_CALL(hipdnnCreateTensorDescriptor(getOutputDescriptor())); } 
MWReLULayer::~MWReLULayer() { } void MWReLULayer::createReLULayer(MWCNNLayer* 
jaqKGCwoANNDMHgAsehk) { setTop(jaqKGCwoANNDMHgAsehk); 
setBatchSize(getTop()->getBatchSize()); setHeight(getTop()->getHeight()); 
setWidth(getTop()->getWidth()); 
setNumInputFeatures(getTop()->getNumOutputFeatures()); 
setNumOutputFeatures(getNumInputFeatures()); 
setWorkSpaceSize(*getTop()->getWorkSpaceSize()); 
CUDNN_CALL(hipdnnSetActivationDescriptor(jscBrjkVJyVfMMDjFpgl, 
HIPDNN_ACTIVATION_RELU, HIPDNN_NOT_PROPAGATE_NAN, 0)); 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(*getOutputDescriptor(), 
HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, getBatchSize(), getNumOutputFeatures(), 
getHeight(), getWidth())); setData(getTop()->getData()); return; } void 
MWReLULayer::predict() { CUDNN_CALL(hipdnnActivationForward(*getCudnnHandle(), 
jscBrjkVJyVfMMDjFpgl, getOnePtr(), *getTop()->getOutputDescriptor(), 
getTop()->getData(), getZeroPtr(), *getOutputDescriptor(),getData())); return; 
} void MWReLULayer::cleanup() { 
CUDNN_CALL(hipdnnDestroyActivationDescriptor(jscBrjkVJyVfMMDjFpgl)); 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(*getOutputDescriptor())); return; } 
MWNormLayer::MWNormLayer() { 
CUDNN_CALL(hipdnnCreateLRNDescriptor(&fYaOQTeunPwVjnhhTECh)); 
CUDNN_CALL(hipdnnCreateTensorDescriptor(getOutputDescriptor())); } 
MWNormLayer::~MWNormLayer() { } void MWNormLayer::createNormLayer(MWCNNLayer* 
jaqKGCwoANNDMHgAsehk, unsigned IwKnaBoXVubIRYcxEJLH, double 
AHqhysOOIgbDpWZoPUFT, double ATEikvMQPqBefhJzjzhc, double CZNYmBcNFSZWvaCklqeM) { 
setTop(jaqKGCwoANNDMHgAsehk); setBatchSize(getTop()->getBatchSize()); 
setHeight(getTop()->getHeight()); setWidth(getTop()->getWidth()); 
setNumInputFeatures(getTop()->getNumOutputFeatures()); 
setNumOutputFeatures(getNumInputFeatures()); 
setWorkSpaceSize(*getTop()->getWorkSpaceSize()); 
CUDNN_CALL(hipdnnSetLRNDescriptor(fYaOQTeunPwVjnhhTECh, 
IwKnaBoXVubIRYcxEJLH, AHqhysOOIgbDpWZoPUFT, ATEikvMQPqBefhJzjzhc, 
CZNYmBcNFSZWvaCklqeM)); CUDNN_CALL(hipdnnSetTensor4dDescriptor(*getOutputDescriptor(), 
HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, getBatchSize(), getNumOutputFeatures(), 
getHeight(), getWidth())); CUDA_CALL(hipMalloc((void**)getDataPtr(), 
sizeof(float)*getBatchSize()*getNumOutputFeatures()*getHeight()*getWidth())); 
return; } void MWNormLayer::predict() { 
CUDNN_CALL(hipdnnLRNCrossChannelForward(*getCudnnHandle(), fYaOQTeunPwVjnhhTECh, 
HIPDNN_LRN_CROSS_CHANNEL, getOnePtr(), *getTop()->getOutputDescriptor(), 
getTop()->getData(), getZeroPtr(), *getOutputDescriptor(),getData())); return; 
} void MWNormLayer::cleanup() { 
CUDNN_CALL(hipdnnDestroyLRNDescriptor(fYaOQTeunPwVjnhhTECh)); 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(*getOutputDescriptor())); if 
(getData()) { call_cuda_free(getData()); } return; } 
MWAvgPoolingLayer::MWAvgPoolingLayer() { 
CUDNN_CALL(hipdnnCreatePoolingDescriptor(&jLyhrFjMmVnNjoeDJCwH)); 
CUDNN_CALL(hipdnnCreateTensorDescriptor(getOutputDescriptor())); } 
MWAvgPoolingLayer::~MWAvgPoolingLayer() { } void 
MWAvgPoolingLayer::createAvgPoolingLayer(MWCNNLayer* jaqKGCwoANNDMHgAsehk, 
int GsZlHFuhbvjLtRMDjXnW, int HJHXkKmgFxxIOsIvRRnF, int HtQBsWTCGEkpylRklilw, int 
IAlDgIFcchbwRGBSfVfA, int GFienSVKLlDQuZeqAdLC, int GeeOVBfQrpMacIFBLKOo) { int 
eqOmMKQRpqBqRQCnJmxt, NldNILHvuQqQPSAHXxdT, XVcMnvCXvZpKICKIjgZi, ugnnrhsgTeWucrMPCJUc; 
setTop(jaqKGCwoANNDMHgAsehk); setBatchSize(getTop()->getBatchSize()); 
setNumInputFeatures(getTop()->getNumOutputFeatures()); 
setNumOutputFeatures(getNumInputFeatures()); 
setWorkSpaceSize(*getTop()->getWorkSpaceSize()); 
CUDNN_CALL(hipdnnSetPooling2dDescriptor(jLyhrFjMmVnNjoeDJCwH, 
HIPDNN_POOLING_AVERAGE_COUNT_INCLUDE_PADDING, HIPDNN_NOT_PROPAGATE_NAN, 
GsZlHFuhbvjLtRMDjXnW, HJHXkKmgFxxIOsIvRRnF, GFienSVKLlDQuZeqAdLC, GeeOVBfQrpMacIFBLKOo, 
HtQBsWTCGEkpylRklilw, IAlDgIFcchbwRGBSfVfA)); 
CUDNN_CALL(hipdnnGetPooling2dForwardOutputDim(jLyhrFjMmVnNjoeDJCwH, 
*getTop()->getOutputDescriptor(), &eqOmMKQRpqBqRQCnJmxt ,&NldNILHvuQqQPSAHXxdT, &XVcMnvCXvZpKICKIjgZi, 
&ugnnrhsgTeWucrMPCJUc)); CUDNN_CALL(hipdnnSetTensor4dDescriptor(*getOutputDescriptor(), 
HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, eqOmMKQRpqBqRQCnJmxt, NldNILHvuQqQPSAHXxdT, XVcMnvCXvZpKICKIjgZi, 
ugnnrhsgTeWucrMPCJUc)); CUDA_CALL(hipMalloc((void**)getDataPtr(), 
sizeof(float)*eqOmMKQRpqBqRQCnJmxt*NldNILHvuQqQPSAHXxdT*XVcMnvCXvZpKICKIjgZi*ugnnrhsgTeWucrMPCJUc)); 
setHeight(XVcMnvCXvZpKICKIjgZi); setWidth(ugnnrhsgTeWucrMPCJUc); return; } void 
MWAvgPoolingLayer::predict() { 
CUDNN_CALL(hipdnnPoolingForward(*getCudnnHandle(), jLyhrFjMmVnNjoeDJCwH, 
getOnePtr(), *getTop()->getOutputDescriptor(), getTop()->getData(), 
getZeroPtr(), *getOutputDescriptor(),getData())); return; } void 
MWAvgPoolingLayer::cleanup() {  
CUDNN_CALL(hipdnnDestroyPoolingDescriptor(jLyhrFjMmVnNjoeDJCwH)); 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(*getOutputDescriptor())); if(getData()) 
{ call_cuda_free(getData()); } return; } MWMaxPoolingLayer::MWMaxPoolingLayer() 
{ CUDNN_CALL(hipdnnCreatePoolingDescriptor(&jLyhrFjMmVnNjoeDJCwH)); 
CUDNN_CALL(hipdnnCreateTensorDescriptor(getOutputDescriptor())); } 
MWMaxPoolingLayer::~MWMaxPoolingLayer() { } void 
MWMaxPoolingLayer::createMaxPoolingLayer(MWCNNLayer* jaqKGCwoANNDMHgAsehk, 
int GsZlHFuhbvjLtRMDjXnW, int HJHXkKmgFxxIOsIvRRnF, int HtQBsWTCGEkpylRklilw, int 
IAlDgIFcchbwRGBSfVfA, int GFienSVKLlDQuZeqAdLC, int GeeOVBfQrpMacIFBLKOo) { int 
eqOmMKQRpqBqRQCnJmxt, NldNILHvuQqQPSAHXxdT, XVcMnvCXvZpKICKIjgZi, ugnnrhsgTeWucrMPCJUc; 
setTop(jaqKGCwoANNDMHgAsehk); setBatchSize(getTop()->getBatchSize()); 
setNumInputFeatures(getTop()->getNumOutputFeatures()); 
setNumOutputFeatures(getNumInputFeatures()); 
setWorkSpaceSize(*getTop()->getWorkSpaceSize()); 
CUDNN_CALL(hipdnnSetPooling2dDescriptor(jLyhrFjMmVnNjoeDJCwH, HIPDNN_POOLING_MAX, 
HIPDNN_NOT_PROPAGATE_NAN, GsZlHFuhbvjLtRMDjXnW, HJHXkKmgFxxIOsIvRRnF, 
GFienSVKLlDQuZeqAdLC, GeeOVBfQrpMacIFBLKOo, HtQBsWTCGEkpylRklilw, 
IAlDgIFcchbwRGBSfVfA)); 
CUDNN_CALL(hipdnnGetPooling2dForwardOutputDim(jLyhrFjMmVnNjoeDJCwH, 
*getTop()->getOutputDescriptor(), &eqOmMKQRpqBqRQCnJmxt ,&NldNILHvuQqQPSAHXxdT, &XVcMnvCXvZpKICKIjgZi, 
&ugnnrhsgTeWucrMPCJUc)); CUDNN_CALL(hipdnnSetTensor4dDescriptor(*getOutputDescriptor(), 
HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, eqOmMKQRpqBqRQCnJmxt, NldNILHvuQqQPSAHXxdT, XVcMnvCXvZpKICKIjgZi, 
ugnnrhsgTeWucrMPCJUc)); CUDA_CALL(hipMalloc((void**)getDataPtr(), 
sizeof(float)*eqOmMKQRpqBqRQCnJmxt*NldNILHvuQqQPSAHXxdT*XVcMnvCXvZpKICKIjgZi*ugnnrhsgTeWucrMPCJUc)); 
setHeight(XVcMnvCXvZpKICKIjgZi); setWidth(ugnnrhsgTeWucrMPCJUc); return; } void 
MWMaxPoolingLayer::predict() { 
CUDNN_CALL(hipdnnPoolingForward(*getCudnnHandle(), jLyhrFjMmVnNjoeDJCwH, 
getOnePtr(), *getTop()->getOutputDescriptor(), getTop()->getData(), 
getZeroPtr(), *getOutputDescriptor(),getData())); return; } void 
MWMaxPoolingLayer::cleanup() { 
CUDNN_CALL(hipdnnDestroyPoolingDescriptor(jLyhrFjMmVnNjoeDJCwH)); 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(*getOutputDescriptor())); if 
(getData()) {  call_cuda_free(getData()); } return; } MWFCLayer::MWFCLayer() : 
ugnnrhsgTeWucrMPCJUc(NULL) , KHClOltUSuqFVVErSxVb(NULL) { 
CUDNN_CALL(hipdnnCreateTensorDescriptor(getOutputDescriptor())); 
CUDNN_CALL(hipdnnCreateTensorDescriptor(&MCrRCXUsCsGPMgQbvMOt)); } 
MWFCLayer::~MWFCLayer() { } void MWFCLayer::createFCLayer(MWCNNLayer* 
jaqKGCwoANNDMHgAsehk, int CGbFsczkgkhjcHoCKzBx, int FrpxvsDMwwgbpqHXWxmN) 
{ setTop(jaqKGCwoANNDMHgAsehk); setBatchSize(getTop()->getBatchSize()); 
setHeight(1); setWidth(1); AzTsxYcYjIEJsGQbeYHm = getTop()->getHeight(); 
BLjrjqvCcCommiXWQLjs = getTop()->getWidth(); 
setNumInputFeatures(CGbFsczkgkhjcHoCKzBx); 
setNumOutputFeatures(FrpxvsDMwwgbpqHXWxmN); 
setWorkSpaceSize(*getTop()->getWorkSpaceSize()); 
CUDA_CALL(hipMalloc((void**)&ugnnrhsgTeWucrMPCJUc, 
sizeof(float)*getNumInputFeatures()*getNumOutputFeatures())); 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(*getOutputDescriptor(), 
HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, getTop()->getBatchSize(), 
getNumOutputFeatures(), 1, 1)); CUDA_CALL(hipMalloc((void**)getDataPtr(), 
sizeof(float)*getBatchSize()*getNumOutputFeatures())); 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(MCrRCXUsCsGPMgQbvMOt, HIPDNN_TENSOR_NCHW, 
HIPDNN_DATA_FLOAT, 1, getNumOutputFeatures(), 1, 1)); 
CUDA_CALL(hipMalloc((void**)&KHClOltUSuqFVVErSxVb, sizeof(float)*getNumOutputFeatures()));
#ifdef RANDOM
 hiprandGenerateNormal(VCbcPxtPsBLTrHYdEvqn, ugnnrhsgTeWucrMPCJUc, 
getNumInputFeatures()*getNumOutputFeatures(), 0, 0.1); 
hiprandGenerateNormal(VCbcPxtPsBLTrHYdEvqn, KHClOltUSuqFVVErSxVb, 
getNumOutputFeatures(), -0.5, 1);
#endif
 return; } void MWFCLayer::loadWeights(const char* SDWKEQTZaTFZByPlzUDR) {  
FILE* UKtMXCCqdjeyaVHabkxg = openBinaryFile(SDWKEQTZaTFZByPlzUDR); 
assert(UKtMXCCqdjeyaVHabkxg); int fhikqqlnUKCjleVKDqiG = 
getNumInputFeatures()*getNumOutputFeatures();  float* NDjzAZSYJuWymuKDNZYB = 
(float*)malloc(sizeof(float)*fhikqqlnUKCjleVKDqiG); fread(NDjzAZSYJuWymuKDNZYB, 
sizeof(float), fhikqqlnUKCjleVKDqiG, UKtMXCCqdjeyaVHabkxg); if( AzTsxYcYjIEJsGQbeYHm != 1 
&& BLjrjqvCcCommiXWQLjs != 1 ) { float* NMMfJylfQjiIUAKhXCJb = 
(float*)malloc(sizeof(float)*AzTsxYcYjIEJsGQbeYHm*BLjrjqvCcCommiXWQLjs); 
for(int k=0; k<fhikqqlnUKCjleVKDqiG/AzTsxYcYjIEJsGQbeYHm/BLjrjqvCcCommiXWQLjs; 
k++) { for(int i=0; i<AzTsxYcYjIEJsGQbeYHm*BLjrjqvCcCommiXWQLjs; i++) 
NMMfJylfQjiIUAKhXCJb[i]=NDjzAZSYJuWymuKDNZYB[k*AzTsxYcYjIEJsGQbeYHm*BLjrjqvCcCommiXWQLjs+i]; 
for(int j=0; j<AzTsxYcYjIEJsGQbeYHm; j++) for(int i=0; 
i<BLjrjqvCcCommiXWQLjs; i++) 
NDjzAZSYJuWymuKDNZYB[k*AzTsxYcYjIEJsGQbeYHm*BLjrjqvCcCommiXWQLjs+j*BLjrjqvCcCommiXWQLjs+i]=NMMfJylfQjiIUAKhXCJb[j+i*AzTsxYcYjIEJsGQbeYHm]; 
} free(NMMfJylfQjiIUAKhXCJb); } CUDA_CALL(hipMemcpy(ugnnrhsgTeWucrMPCJUc, 
NDjzAZSYJuWymuKDNZYB, sizeof(float)*fhikqqlnUKCjleVKDqiG, hipMemcpyHostToDevice));
#if 0
 printf("%s loaded. Size = %d. %f\n", SDWKEQTZaTFZByPlzUDR, fhikqqlnUKCjleVKDqiG, NDjzAZSYJuWymuKDNZYB[0]);
#endif
 free(NDjzAZSYJuWymuKDNZYB); fclose(UKtMXCCqdjeyaVHabkxg); return; } void 
MWFCLayer::loadBias(const char* SDWKEQTZaTFZByPlzUDR) { FILE* UKtMXCCqdjeyaVHabkxg = 
openBinaryFile(SDWKEQTZaTFZByPlzUDR); assert(UKtMXCCqdjeyaVHabkxg); int fhikqqlnUKCjleVKDqiG 
= getNumOutputFeatures();  float* NDjzAZSYJuWymuKDNZYB = 
(float*)malloc(sizeof(float)*fhikqqlnUKCjleVKDqiG); fread(NDjzAZSYJuWymuKDNZYB, 
sizeof(float), fhikqqlnUKCjleVKDqiG, UKtMXCCqdjeyaVHabkxg); 
CUDA_CALL(hipMemcpy(KHClOltUSuqFVVErSxVb, NDjzAZSYJuWymuKDNZYB, 
sizeof(float)*fhikqqlnUKCjleVKDqiG, hipMemcpyHostToDevice)); 
free(NDjzAZSYJuWymuKDNZYB); fclose(UKtMXCCqdjeyaVHabkxg); return; } void 
MWFCLayer::predict() { int aLsOwwcceEmRSYzllBNs=1; int bDTIjtxZiSHtjwzgEluE=1; if( 
getBatchSize()==1 ) { CUDA_CALL(hipMemcpy(getData(), KHClOltUSuqFVVErSxVb, 
sizeof(float)*getNumOutputFeatures(), hipMemcpyDeviceToDevice)); 
CUBLAS_CALL(hipblasSgemv(*getCublasHandle(), HIPBLAS_OP_T, getNumInputFeatures(), 
getNumOutputFeatures(), getOnePtr(), ugnnrhsgTeWucrMPCJUc, getNumInputFeatures(), 
getTop()->getData(), aLsOwwcceEmRSYzllBNs, getOnePtr(),getData(), bDTIjtxZiSHtjwzgEluE)); 
} else { CUBLAS_CALL(hipblasSgemm(*getCublasHandle(), HIPBLAS_OP_T, HIPBLAS_OP_N, 
getNumOutputFeatures(), getBatchSize(), getNumInputFeatures(), getOnePtr(), 
ugnnrhsgTeWucrMPCJUc, getNumInputFeatures(), getTop()->getData(), 
getNumInputFeatures(), getZeroPtr(),getData(), getNumOutputFeatures())); 
CUDNN_CALL(hipdnnAddTensor(*getCudnnHandle(), getOnePtr(), MCrRCXUsCsGPMgQbvMOt, 
KHClOltUSuqFVVErSxVb, getOnePtr(), *getOutputDescriptor(),getData())); } return; } void 
MWFCLayer::cleanup() { if (ugnnrhsgTeWucrMPCJUc) { call_cuda_free(ugnnrhsgTeWucrMPCJUc); }  
CUDNN_CALL(hipdnnDestroyTensorDescriptor(MCrRCXUsCsGPMgQbvMOt)); if 
(KHClOltUSuqFVVErSxVb) { call_cuda_free(KHClOltUSuqFVVErSxVb); } 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(*getOutputDescriptor())); if 
(getData()) { call_cuda_free(getData()); } return; } 
MWSoftmaxLayer::MWSoftmaxLayer() { 
CUDNN_CALL(hipdnnCreateTensorDescriptor(getOutputDescriptor())); } 
MWSoftmaxLayer::~MWSoftmaxLayer() { } void 
MWSoftmaxLayer::createSoftmaxLayer(MWCNNLayer* jaqKGCwoANNDMHgAsehk) { 
setTop(jaqKGCwoANNDMHgAsehk); setBatchSize(getTop()->getBatchSize()); 
setHeight(getTop()->getHeight()); setWidth(getTop()->getWidth()); 
setNumInputFeatures(getTop()->getNumOutputFeatures()); 
setNumOutputFeatures(getNumInputFeatures()); 
setWorkSpaceSize(*getTop()->getWorkSpaceSize()); 
CUDNN_CALL(hipdnnSetTensor4dDescriptor(*getOutputDescriptor(), 
HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, getBatchSize(), getNumOutputFeatures(), 
getHeight(), getWidth())); CUDA_CALL(hipMalloc((void**)getDataPtr(), 
sizeof(float)*getBatchSize()*getNumInputFeatures()*getHeight()*getWidth())); 
return; } void MWSoftmaxLayer::predict() { 
CUDNN_CALL(hipdnnSoftmaxForward(*getCudnnHandle(), HIPDNN_SOFTMAX_ACCURATE, 
HIPDNN_SOFTMAX_MODE_CHANNEL, getOnePtr(), *getTop()->getOutputDescriptor(), 
getTop()->getData(), getZeroPtr(), *getOutputDescriptor(),getData())); return; 
} void MWSoftmaxLayer::cleanup() { 
CUDNN_CALL(hipdnnDestroyTensorDescriptor(*getOutputDescriptor())); if 
(getData()) {  call_cuda_free(getData()); } return; } 
MWOutputLayer::MWOutputLayer() { } MWOutputLayer::~MWOutputLayer() { } void 
MWOutputLayer::createOutputLayer(MWCNNLayer* jaqKGCwoANNDMHgAsehk) { 
setTop(jaqKGCwoANNDMHgAsehk); setBatchSize(getTop()->getBatchSize()); 
setHeight(getTop()->getHeight()); setWidth(getTop()->getWidth()); 
setNumInputFeatures(getTop()->getNumOutputFeatures()); 
setNumOutputFeatures(getNumInputFeatures()); 
setWorkSpaceSize(*getTop()->getWorkSpaceSize()); setData(getTop()->getData()); 
return; } void MWOutputLayer::predict() { return; } void 
MWOutputLayer::cleanup() { return; } MWPassthroughLayer::MWPassthroughLayer() { 
} MWPassthroughLayer::~MWPassthroughLayer() { } void 
MWPassthroughLayer::createPassthroughLayer(MWCNNLayer* 
jaqKGCwoANNDMHgAsehk) { setTop(jaqKGCwoANNDMHgAsehk); 
setBatchSize(getTop()->getBatchSize()); setHeight(getTop()->getHeight()); 
setWidth(getTop()->getWidth()); 
setNumInputFeatures(getTop()->getNumOutputFeatures()); 
setNumOutputFeatures(getNumInputFeatures()); 
setWorkSpaceSize(*getTop()->getWorkSpaceSize()); setData(getTop()->getData()); 
return; } void MWPassthroughLayer::predict() { return; } void 
MWPassthroughLayer::cleanup() { return; }