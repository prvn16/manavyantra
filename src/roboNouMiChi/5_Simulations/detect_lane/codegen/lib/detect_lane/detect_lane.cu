#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: detect_lane.cu
//
// GPU Coder version                    : 1.0
// CUDA/C/C++ source code generated on  : 07-Jan-2018 02:03:56
//

// Include Files
#include "rt_nonfinite.h"
#include "detect_lane.h"
#include "DeepLearningNetwork.h"

// Variable Definitions
static c_lanenet b_lanenet;
static boolean_T lanenet_not_empty;

// Function Definitions

//
// From the networks output,
//  compute left and right lane points in the image coordinates
//  The camera coordinates are described by the caltech mono camera model.
// Arguments    : const real32_T frame[154587]
//                const real_T laneCoeffMeans[6]
//                const real_T laneCoeffStds[6]
//                boolean_T *laneFound
//                real32_T ltPts[56]
//                real32_T rtPts[56]
// Return Type  : void
//
void detect_lane(const real32_T frame[154587], const real_T laneCoeffMeans[6],
                 const real_T laneCoeffStds[6], boolean_T *laneFound, real32_T
                 ltPts[56], real32_T rtPts[56])
{
  uint8_T outsz[3];
  static real32_T b[154587];
  int32_T iwork[3];
  int32_T i0;
  int32_T k;
  int32_T inc[3];
  static const int8_T iv0[3] = { 1, 0, 2 };

  int32_T idest;
  int32_T isrc;
  real32_T lanecoeffsNetworkOutput[6];
  real32_T rt_y[28];
  real32_T lt_y[28];
  real32_T X[84];
  real32_T varargin_1[56];
  real32_T U[84];
  static const real32_T fv0[9] = { 309.430603F, -309.436188F, 168.170944F,
    166.611893F, -1.02020358E-14F, 863.843445F, 0.970295727F, -5.94134783E-17F,
    0.527341366F };

  int32_T initAuxVar;
  int32_T exitg1;

  // METHOD1 Summary of this method goes here
  //    Detailed explanation goes here
  if (!lanenet_not_empty) {
    DeepLearningNetwork_setup(&b_lanenet);
    lanenet_not_empty = true;
  }

  outsz[0] = 227;
  outsz[1] = 227;
  outsz[2] = 3;
  for (i0 = 0; i0 < 3; i0++) {
    iwork[i0] = 1;
  }

  for (k = 0; k < 2; k++) {
    iwork[k + 1] = iwork[k] * 227;
  }

  for (i0 = 0; i0 < 3; i0++) {
    inc[i0] = iwork[iv0[i0]];
  }

  for (i0 = 0; i0 < 3; i0++) {
    iwork[i0] = 0;
  }

  idest = 0;
  do {
    isrc = 0;
    for (k = 0; k < 2; k++) {
      isrc += iwork[k + 1] * inc[k + 1];
    }

    initAuxVar = idest;
    for (k = 0; k < 227; k++) {
      idest = initAuxVar + k;
      b[idest] = frame[isrc];
      idest++;
      isrc += inc[0];
    }

    k = 1;
    do {
      exitg1 = 0;
      iwork[k]++;
      if (iwork[k] < outsz[k]) {
        exitg1 = 2;
      } else if (k + 1 == 3) {
        exitg1 = 1;
      } else {
        iwork[1] = 0;
        k = 2;
      }
    } while (exitg1 == 0);
  } while (!(exitg1 == 1));

  DeepLearningNetwork_predict(&b_lanenet, b, lanecoeffsNetworkOutput);

  //  Recover original coeffs by reversing the normalization steps
  for (i0 = 0; i0 < 6; i0++) {
    lanecoeffsNetworkOutput[i0] = lanecoeffsNetworkOutput[i0] * (real32_T)
      laneCoeffStds[i0] + (real32_T)laneCoeffMeans[i0];
  }

  // c should be more than 0.5 for it to be a right lane
  // meters, ahead of the sensor
  if ((std::abs(lanecoeffsNetworkOutput[5]) > 0.5F) && (std::abs
       (lanecoeffsNetworkOutput[2]) > 6.5F)) {
    for (i0 = 0; i0 < 28; i0++) {
      rt_y[i0] = lanecoeffsNetworkOutput[3];
    }

    for (k = 0; k < 2; k++) {
      for (i0 = 0; i0 < 28; i0++) {
        rt_y[i0] = (3.0F + (real32_T)i0) * rt_y[i0] + lanecoeffsNetworkOutput[k
          + 4];
      }
    }

    for (i0 = 0; i0 < 28; i0++) {
      lt_y[i0] = lanecoeffsNetworkOutput[0];
    }

    for (k = 0; k < 2; k++) {
      for (i0 = 0; i0 < 28; i0++) {
        lt_y[i0] = (3.0F + (real32_T)i0) * lt_y[i0] + lanecoeffsNetworkOutput[k
          + 1];
      }
    }

    //  Visualize lane boundaries of the ego vehicle
    //  map vehicle to image coordinates
    for (idest = 0; idest < 28; idest++) {
      varargin_1[idest] = 3.0F + (real32_T)idest;
      varargin_1[28 + idest] = lt_y[idest];
      X[56 + idest] = 1.0F;
    }

    for (isrc = 0; isrc < 2; isrc++) {
      for (idest = 0; idest < 28; idest++) {
        X[idest + 28 * isrc] = varargin_1[idest + 28 * isrc];
      }
    }

    for (i0 = 0; i0 < 28; i0++) {
      for (idest = 0; idest < 3; idest++) {
        U[i0 + 28 * idest] = 0.0F;
        for (isrc = 0; isrc < 3; isrc++) {
          U[i0 + 28 * idest] += X[i0 + 28 * isrc] * fv0[isrc + 3 * idest];
        }
      }
    }

    for (idest = 0; idest < 2; idest++) {
      isrc = idest * 28;
      for (k = 0; k < 28; k++) {
        varargin_1[isrc + k] = U[56 + k];
      }
    }

    for (i0 = 0; i0 < 2; i0++) {
      for (idest = 0; idest < 28; idest++) {
        U[idest + 28 * i0] /= varargin_1[idest + 28 * i0];
        ltPts[idest + 28 * i0] = U[idest + 28 * i0];
      }
    }

    for (idest = 0; idest < 28; idest++) {
      varargin_1[idest] = 3.0F + (real32_T)idest;
      varargin_1[28 + idest] = rt_y[idest];
      X[56 + idest] = 1.0F;
    }

    for (isrc = 0; isrc < 2; isrc++) {
      for (idest = 0; idest < 28; idest++) {
        X[idest + 28 * isrc] = varargin_1[idest + 28 * isrc];
      }
    }

    for (i0 = 0; i0 < 28; i0++) {
      for (idest = 0; idest < 3; idest++) {
        U[i0 + 28 * idest] = 0.0F;
        for (isrc = 0; isrc < 3; isrc++) {
          U[i0 + 28 * idest] += X[i0 + 28 * isrc] * fv0[isrc + 3 * idest];
        }
      }
    }

    for (idest = 0; idest < 2; idest++) {
      isrc = idest * 28;
      for (k = 0; k < 28; k++) {
        varargin_1[isrc + k] = U[56 + k];
      }
    }

    for (i0 = 0; i0 < 2; i0++) {
      for (idest = 0; idest < 28; idest++) {
        U[idest + 28 * i0] /= varargin_1[idest + 28 * i0];
        rtPts[idest + 28 * i0] = U[idest + 28 * i0];
      }
    }

    *laneFound = true;
  } else {
    *laneFound = false;
  }
}

//
// Arguments    : void
// Return Type  : void
//
void detect_lane_init()
{
  lanenet_not_empty = false;
}

//
// File trailer for detect_lane.cu
//
// [EOF]
//
