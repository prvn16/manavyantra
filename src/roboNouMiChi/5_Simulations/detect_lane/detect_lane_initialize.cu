//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: detect_lane_initialize.cu
//
// GPU Coder version                    : 1.0
// CUDA/C/C++ source code generated on  : 07-Jan-2018 02:03:56
//

// Include Files
#include "rt_nonfinite.h"
#include "detect_lane.h"
#include "detect_lane_initialize.h"

// Function Definitions

//
// Arguments    : void
// Return Type  : void
//
void detect_lane_initialize()
{
  rt_InitInfAndNaN(8U);
  detect_lane_init();
}

//
// File trailer for detect_lane_initialize.cu
//
// [EOF]
//
