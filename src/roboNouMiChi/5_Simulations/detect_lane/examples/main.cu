//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: main.cu
//
// GPU Coder version                    : 1.0
// CUDA/C/C++ source code generated on  : 07-Jan-2018 02:03:56
//

//***********************************************************************
// This automatically generated example C main file shows how to call
// entry-point functions that MATLAB Coder generated. You must customize
// this file for your application. Do not modify this file directly.
// Instead, make a copy of this file, modify it, and integrate it into
// your development environment.
//
// This file initializes entry-point function arguments to a default
// size and value before calling the entry-point functions. It does
// not store or use any values returned from the entry-point functions.
// If necessary, it does pre-allocate memory for returned values.
// You can use this file as a starting point for a main function that
// you can deploy in your application.
//
// After you copy the file, and before you deploy it, you must make the
// following changes:
// * For variable-size function arguments, change the example sizes to
// the sizes that your application requires.
// * Change the example values of function arguments to the values that
// your application requires.
// * If the entry-point functions return values, store these values or
// otherwise use them as required by your application.
//
//***********************************************************************
// Include Files
#include "rt_nonfinite.h"
#include "detect_lane.h"
#include "main.h"
#include "detect_lane_terminate.h"
#include "detect_lane_initialize.h"

// Function Declarations
static void argInit_1x6_real_T(real_T result[6]);
static void argInit_227x227x3_real32_T(real32_T result[154587]);
static real32_T argInit_real32_T();
static real_T argInit_real_T();
static void main_detect_lane();

// Function Definitions

//
// Arguments    : real_T result[6]
// Return Type  : void
//
static void argInit_1x6_real_T(real_T result[6])
{
  int32_T idx1;

  // Loop over the array to initialize each element.
  for (idx1 = 0; idx1 < 6; idx1++) {
    // Set the value of the array element.
    // Change this value to the value that the application requires.
    result[idx1] = argInit_real_T();
  }
}

//
// Arguments    : real32_T result[154587]
// Return Type  : void
//
static void argInit_227x227x3_real32_T(real32_T result[154587])
{
  int32_T idx0;
  int32_T idx1;
  int32_T idx2;

  // Loop over the array to initialize each element.
  for (idx0 = 0; idx0 < 227; idx0++) {
    for (idx1 = 0; idx1 < 227; idx1++) {
      for (idx2 = 0; idx2 < 3; idx2++) {
        // Set the value of the array element.
        // Change this value to the value that the application requires.
        result[(idx0 + 227 * idx1) + 51529 * idx2] = argInit_real32_T();
      }
    }
  }
}

//
// Arguments    : void
// Return Type  : real32_T
//
static real32_T argInit_real32_T()
{
  return 0.0F;
}

//
// Arguments    : void
// Return Type  : real_T
//
static real_T argInit_real_T()
{
  return 0.0;
}

//
// Arguments    : void
// Return Type  : void
//
static void main_detect_lane()
{
  boolean_T laneFound;
  real32_T ltPts[56];
  real32_T rtPts[56];
  static real32_T b[154587];
  real_T c[6];
  real_T d[6];

  // Initialize function 'detect_lane' input arguments.
  // Initialize function input argument 'frame'.
  // Initialize function input argument 'laneCoeffMeans'.
  // Initialize function input argument 'laneCoeffStds'.
  // Call the entry-point 'detect_lane'.
  argInit_227x227x3_real32_T(b);
  argInit_1x6_real_T(c);
  argInit_1x6_real_T(d);
  detect_lane(b, c, d, &laneFound, ltPts, rtPts);
}

//
// Arguments    : int32_T argc
//                const char * const argv[]
// Return Type  : int32_T
//
int32_T main(int32_T, const char * const [])
{
  // Initialize the application.
  // You do not need to do this more than one time.
  detect_lane_initialize();

  // Invoke the entry-point functions.
  // You can call entry-point functions multiple times.
  main_detect_lane();

  // Terminate the application.
  // You do not need to do this more than one time.
  detect_lane_terminate();
  return 0;
}

//
// File trailer for main.cu
//
// [EOF]
//
